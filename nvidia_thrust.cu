#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <string>
#include <algorithm>
#include <math.h>
#include <map>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/complex.h>
#include <thrust/sort.h>

#define NODE_NUM 1850065 // Number of nodes in the graph. 1849272

#define ALPHA 0.2        // Dumping factor
#define EPSILON 0.000001 // Very small number for convergence

#define GRAPH "graph.txt"

using namespace std;

// fabs function is turned to a functor in order to use it with thrust transform.
struct absoluteValue {
    __host__ __device__
    double operator()(const double& a, const double& b){
        return fabs(a - b);
    }    
};

int main()
{
    thrust::host_vector<int> row_begin;
    thrust::host_vector<double> values;
    thrust::host_vector<int> col_indices;

    vector<pair<string, string>> graph;

    int nodeIndex = 0;            // To use it to assign indexes to nodes.
    vector<int> out(NODE_NUM);    // Outgoing Degrees of the nodes.
    map<string, int> nodeIndexes; // Hash map of names and indexes of the nodes.
    int i = 0;
    for (i = 0; i < NODE_NUM; i++)
    {
        out.at(i) = 0;
    }

    ifstream file(GRAPH);

    cout << "Opening the file..." << endl;
    if (!file.is_open())
    {
        cout << "Failed to open file.\n";
        return 0;
    }

    cout << "File is opened. Reading..." << endl;

    string line;
    string node1;
    string node2;

    while (getline(file, line)) // Stores graph in a vector to iterate in it faster afterwards.
    {
        istringstream ss(line);

        ss >> node1 >> node2;
        graph.emplace_back(node1, node2);
    }
    file.close();

    cout << "File has been read." << endl;
    cout << "Assigning indexes..." << endl;

    for (auto g : graph)
    { // Assignes indexes to the nodes to refer to them afterwards.
        string node1 = g.first;
        string node2 = g.second;

        int node1Index = 0;
        int node2Index = 0;

        auto node1Find = nodeIndexes.find(node1);
        if (node1Find == nodeIndexes.end()) // if node is not given index
        {
            node1Index = nodeIndex;
            nodeIndex++;
            nodeIndexes.emplace(node1, node1Index);
        }
        else
        {
            node1Index = node1Find->second;
        }

        out.at(node1Index)++;

        auto node2Find = nodeIndexes.find(node2);
        if (node2Find == nodeIndexes.end()) // if node is not given index
        {
            node2Index = nodeIndex;
            nodeIndex++;
            nodeIndexes.emplace(node2, node2Index);
        }
    }
    cout << "Indexes are assigned." << endl;

    cout << "Creating CSR Matrix..." << endl;

    string lastNode;
    for (auto g : graph)
    {
        int node1Index = 0;

        string node1 = g.first;
        string node2 = g.second;

        auto node1Find = nodeIndexes.find(node1);

        node1Index = node1Find->second;

        if (node2 != lastNode) // if node2 is not the same with the previous node, this row finishes.
        {
            row_begin.push_back(values.size());
        }

        col_indices.push_back(node1Index);          // column index is equal to index of node1.
        values.push_back(1.0 / out.at(node1Index)); // values are calculated based on outgoing degrees of nodes.

        lastNode = node2;
    }

    row_begin.push_back(values.size()); // For the last row

    cout << "CSR Matrix is created" << endl;

    cout << "Starting rank calculations..." << endl;

    int iterationCount = 0;
    double diff;
    thrust::host_vector<double> ranks(NODE_NUM, 1.0);    // r^t
    thrust::host_vector<double> newRanks(NODE_NUM, 0); // r^t+1
    thrust::host_vector<double> diffs(NODE_NUM); // differences between each rank in r^t, r^t+1
    
    double start = clock(); // Start time

    while (true)
    {
        iterationCount++;
        diff = 0.0;
        for (i = 0; i < (int)row_begin.size() - 1; i++)
        {
            int rowStart = row_begin[i]; 
            int rowEnd = row_begin[i+1];
            // cout << "rowStart: " << rowStart << endl;
            // previousRank = r^t -> through rowStart to rowEnd
            auto previousRank = thrust::make_permutation_iterator(ranks.begin(), col_indices.begin() + rowStart);
            // totalAlpha = In each calculation we need to sum with (1.0 - ALPHA). So we need rowEnd - rowStart time of it.
            double totalAlpha = (1.0 - ALPHA) * (rowEnd - rowStart);
            // totalMultiplication = The total of P*r(t) -> t throught rowStart to rowEnd
            double totalMultiplication = thrust::inner_product(values.begin() + rowStart, values.begin() + rowEnd, previousRank, 0.0);
            newRanks[i] = (ALPHA * totalMultiplication) + totalAlpha;
        }

        thrust::transform(newRanks.begin(), newRanks.end(), ranks.begin(), diffs.begin(), absoluteValue());

        diff = thrust::reduce(diffs.begin(), diffs.end());
        cout << "Difference: " << diff << endl;
        
        ranks = newRanks; // ranks are assigned to the newly calculated ranks.
        if (diff < EPSILON)
            break; // Finishes iteration when L1 Norm is smaller than Epsilon value.
    }

    double end = clock(); // End Time

    double timePassed = end - start; // Time passed in calculation

    cout << "Time passed in calculations: " << timePassed << endl;
    cout << "Iteration Count: " << iterationCount << endl;

    cout << "Highest Ranked 5 Nodes: " << endl;

    for (i = 0; i < 5; i++)
    {
        int index = 0;
        double rank = 0;
        string node;

        for (int j = 0; j < NODE_NUM; j++) // Finds highest ranked node index.
        {
            double r = ranks[j];
            if (r > rank)
            {
                index = j;
                rank = r;
            }
        }

        ranks[index] = 0.0; // Node will not be considered in the next iteration

        for (auto it = nodeIndexes.begin(); it != nodeIndexes.end(); it++) // Finds the name of the node with the index
        {
            if (it->second == index)
            {
                node = it->first;
                break;
            }
        }
        cout << i + 1 << "- " << node << " -- " << rank << endl;
    }

    return 1;
}